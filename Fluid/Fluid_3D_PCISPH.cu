#include "hip/hip_runtime.h"
#include "Fluid_3D_PCISPH.cuh"

#define SIMULATE_PARTICLES_NOT_FLUID 0

namespace Fluid_3D_PCISPH {
	// this is not for PCISPH.
	// It is used for a pure particle simulation, same as the one in CUDA samples
	__global__ void collide(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius, float timestep, float spacing) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / cellSize;
		thisCell.y = pos.y / cellSize;
		thisCell.z = pos.z / cellSize;

		float3 force = { 0,0,0 };

		float collideDist = spacing;;

#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {

						if (j != index) {
							Particle& that = particles[j];
							float3 relPos = that.position - particle.position;
							float dist = length(relPos);

							if (dist < collideDist) {
								float3 norm = relPos / dist;

								// relative velocity
								float3 relVel = that.velosity - particle.velosity;

								// relative tangential velocity
								float3 tanVel = relVel - (dot(relVel, norm) * norm);

								// spring force
								force += -0.5 * (collideDist - dist) * norm;
								// dashpot (damping) force
								force += 0.02 * relVel;
								// tangential shear force
								force += 0.1 * tanVel;
							}

						}

					}
				}
			}
		}

		particle.velosity += force;
	}
	// this is not for PCISPH.
	// It is used for a pure particle simulation, same as the one in CUDA samples
	__global__ void integrate(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float3 gridPhysicalSize, float kernelRadius, float timestep, float spacing) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		particle.velosity += make_float3(0, -0.0003, 0) * timestep;

		float3 pos = particle.position;
		float3 vel = particle.velosity;

		pos += timestep * vel;

		float bounce = -0.5;

		if (pos.x < spacing) {
			pos.x = spacing;
			vel.x *= bounce;;
		}

		if (pos.x > gridPhysicalSize.x - spacing) {
			pos.x = gridPhysicalSize.x - spacing;
			vel.x *= bounce;;
		}

		if (pos.y < spacing) {
			pos.y = spacing;
			vel.y *= bounce;;
		}

		if (pos.y > gridPhysicalSize.y - spacing) {
			pos.y = gridPhysicalSize.y - spacing;
			vel.y *= bounce;;
		}

		if (pos.z < spacing) {
			pos.z = spacing;
			vel.z *= bounce;;
		}

		if (pos.z > gridPhysicalSize.z - spacing) {
			pos.z = gridPhysicalSize.z - spacing;
			vel.z *= bounce;;
		}



		particle.position = pos;
		particle.velosity = vel;

	}













	__global__ void computeExternalForcesImpl(Particle* particles, int particleCount) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];
		particle.acceleration = make_float3(0, -9.8, 0);

	}

	__global__ void initPressureImpl(Particle* particles, int particleCount) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];
		particle.pressure = 0;
		particle.pressureForces = make_float3(0, 0, 0);

	}


	__global__ void predictVelocityAndPositionImpl(Particle* particles, int particleCount, float timestep, bool setAsActual, float spacing, float3 gridPhysicalSize) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];

		float3 acc = particle.acceleration + particle.pressureForces;
		float3 vel = particle.velosity + acc * timestep;
		float3 pos = particle.position + vel * timestep;

		float bounce = -0.0;

		float minDistanceFromWall = spacing / 2;

		if (pos.x < minDistanceFromWall) {
			pos.x = minDistanceFromWall;
			vel.x *= bounce;;
		}

		if (pos.x > gridPhysicalSize.x - minDistanceFromWall) {
			pos.x = gridPhysicalSize.x - minDistanceFromWall;
			vel.x *= bounce;;
		}

		if (pos.y < minDistanceFromWall) {
			pos.y = minDistanceFromWall;
			vel.y *= bounce;;
		}

		if (pos.y > gridPhysicalSize.y - minDistanceFromWall) {
			pos.y = gridPhysicalSize.y - minDistanceFromWall;
			vel.y *= bounce;;
		}

		if (pos.z < minDistanceFromWall) {
			pos.z = minDistanceFromWall;
			vel.z *= bounce;;
		}

		if (pos.z > gridPhysicalSize.z - minDistanceFromWall) {
			pos.z = gridPhysicalSize.z - minDistanceFromWall;
			vel.z *= bounce;;
		}

		if (setAsActual) {
			particle.position = pos;
			particle.velosity = vel;

		}
		else {
			particle.predictedPosition = pos;
			particle.predictedVelocity = vel;
		}

	}

	__global__ void predictDensityAndPressureImpl(Particle* particles,int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius,float kernelRadius2,float kernelRadius6,float kernelRadius9, bool setAsRest, float timestep) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / kernelRadius;
		thisCell.y = pos.y / kernelRadius;
		thisCell.z = pos.z / kernelRadius;

		float rho0 = particle.restDensity;

		float beta = timestep * timestep * 2 / (rho0 * rho0);

		float density = 0;

		float3 sumGradW = make_float3(0, 0, 0);
		float sumGradWDot = 0;


#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {
						Particle& that = particles[j];
						float3 posDiff = particle.predictedPosition - that.predictedPosition;
						float thisDensityContribution = poly6(posDiff, kernelRadius2,kernelRadius9);



						density += thisDensityContribution;


						float3 gradW = spikey_grad(posDiff, kernelRadius,kernelRadius6);
						sumGradW += gradW;
						sumGradWDot += dot(gradW, gradW);
					}
				}
			}
		}
		particle.density = density;

		if (setAsRest) {
			particle.restDensity = density;
			
		}


		float rhoError = density - rho0;
		float correctionCoeff = 1.0 / (beta * (dot(sumGradW, sumGradW) + sumGradWDot));

		correctionCoeff = 50.0;

		float pressureCorrection = correctionCoeff * rhoError;
		particle.pressure += pressureCorrection;


	}

	__global__ void computePressureForceImpl(Particle* particles, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius,float kernelRadius2, float kernelRadius6, float kernelRadius9 ) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / kernelRadius;
		thisCell.y = pos.y / kernelRadius;
		thisCell.z = pos.z / kernelRadius;

		float3 force = make_float3(0, 0, 0);

#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {
						Particle that = particles[j];
						force -= spikey_grad(particle.predictedPosition - that.predictedPosition, kernelRadius,kernelRadius6)
							* ((that.pressure / (that.density * that.density)) + (particle.pressure / (particle.density * particle.density)));
					}
				}
			}
		}
		particle.pressureForces = force;

	}







	Fluid::Fluid() {

	}

	void Fluid::draw(const DrawCommand& drawCommand){
		skybox.draw(drawCommand);
		//container.draw(drawCommand);


		if (drawCommand.renderMode == RenderMode::Mesh) {
			hipDeviceSynchronize();
			mesher->mesh(particles, particlesCopy, particleHashes, particleIndices, meshRenderer->coordsDevice);
			hipDeviceSynchronize();
			meshRenderer->draw(drawCommand, skybox.texSkyBox);
		}
		else {
			updatePositionsVBO << <numBlocks, numThreads >> > (particles, pointSprites->positionsDevice, particleCount);
			hipDeviceSynchronize();
			pointSprites->draw(drawCommand, particleSpacing/2, skybox.texSkyBox);
		}

	}

	void Fluid::createSquareFluid(std::vector<Particle>& particlesVec, float3 minPos, float3 maxPos) {
		float minDistanceFromWall = particleSpacing / 2.f;

		float3 minPhysicalPos = {
			minPos.x * gridPhysicalSize.x,
			minPos.y* gridPhysicalSize.y,
			minPos.z* gridPhysicalSize.z,
		};
		minPhysicalPos += make_float3(1, 1, 1) * minDistanceFromWall;
		float3 maxPhysicalPos = {
			maxPos.x* gridPhysicalSize.x,
			maxPos.y* gridPhysicalSize.y,
			maxPos.z* gridPhysicalSize.z,
		};
		maxPhysicalPos -= make_float3(1, 1, 1) * (minDistanceFromWall - 1e-3);
		for (float x = minPhysicalPos.x ; x <= maxPhysicalPos.x; x += particleSpacing) {
			for (float y = minPhysicalPos.y; y <= maxPhysicalPos.y ; y += particleSpacing) {
				for (float z = minPhysicalPos.z; z <= maxPhysicalPos.z ; z += particleSpacing) {
					float jitterMagnitude = particleSpacing/2.f;
					float3 jitter;
					jitter.x = (random0to1() - 0.5);
					jitter.y = (random0to1() - 0.5);
					jitter.z = (random0to1() - 0.5);
					jitter *= jitterMagnitude;
					float3 pos = make_float3(x, y, z);
					pos += jitter;


					
					
					pos.x = min(gridPhysicalSize.x - minDistanceFromWall, max(minDistanceFromWall, pos.x));
					pos.y = min(gridPhysicalSize.y - minDistanceFromWall, max(minDistanceFromWall, pos.y));
					pos.z = min(gridPhysicalSize.z - minDistanceFromWall, max(minDistanceFromWall, pos.z));

					particlesVec.emplace_back(pos);

				}
			}
		}
	}
	void Fluid::createSphereFluid(std::vector<Particle>& particlesVec, float3 center, float radius) {

		float3 minPhysicalPos = {
			0,0,0
		};
		minPhysicalPos += make_float3(1, 1, 1) * particleSpacing * 0.5;
		float3 maxPhysicalPos = gridPhysicalSize;
		maxPhysicalPos -= make_float3(1, 1, 1) * particleSpacing * 0.5;

		float3 physicalCenter = {
			center.x * gridPhysicalSize.x,
			center.y * gridPhysicalSize.y,
			center.z * gridPhysicalSize.z
		};

		float physicalRadius = radius * gridPhysicalSize.y;

		for (float x = minPhysicalPos.x; x < maxPhysicalPos.x; x += particleSpacing) {
			for (float y = minPhysicalPos.y; y < maxPhysicalPos.y; y += particleSpacing) {
				for (float z = minPhysicalPos.z; z < maxPhysicalPos.z; z += particleSpacing) {

					float3 pos = make_float3(x, y, z);
					float3 jitter = make_float3(1, 1, 1);
					jitter.x *= (random0to1() - 0.5)*particleSpacing*0.01;
					jitter.y *= (random0to1() - 0.5) * particleSpacing * 0.01;
					jitter.z *= (random0to1() - 0.5) * particleSpacing * 0.01;

#if  SIMULATE_PARTICLES_NOT_FLUID
					pos += jitter;
#endif //  SIMULATE_PARTICLES_NOT_FLUID


					
					if (length(pos-physicalCenter) < physicalRadius) {
						
						particlesVec.emplace_back(pos);
					}
				}
			}
		}
	}

	void Fluid::init(std::shared_ptr<FluidConfig> config) {

#if SIMULATE_PARTICLES_NOT_FLUID

		kernelRadius = gridPhysicalSize.x / 64;
		particleSpacing = kernelRadius / 2;
		
#else
		particleSpacing = pow(gridPhysicalSize.x * gridPhysicalSize.y * gridPhysicalSize.z / particleCountWhenFull, 1.0 / 3.0);

		particleSpacing = gridPhysicalSize.x / ceil(gridPhysicalSize.x / particleSpacing); // so that gridPhysicalSize is exact multiple.

		kernelRadius = particleSpacing * kernelRadiusToSpacingRatio;
		kernelRadius2 = kernelRadius * kernelRadius;
		kernelRadius6 = kernelRadius2 * kernelRadius2 * kernelRadius2;
		kernelRadius9 = kernelRadius6 * kernelRadius2 * kernelRadius;
#endif

		

		std::vector<Particle> particlesVec;

		std::shared_ptr<FluidConfig3D> config3D = std::static_pointer_cast<FluidConfig3D, FluidConfig>(config);
		for (const InitializationVolume& vol : config3D->initialVolumes) {
			if (vol.shapeType == ShapeType::Square) {
				float3 minPos = make_float3(vol.params[0], vol.params[1], vol.params[2]);
				float3 maxPos = make_float3(vol.params[3], vol.params[4], vol.params[5]);
				createSquareFluid(particlesVec,minPos,maxPos);
			}
			else if (vol.shapeType == ShapeType::Sphere) {
				float3 center = make_float3(vol.params[0], vol.params[1], vol.params[2]);
				float radius = vol.params[3];
				createSphereFluid(particlesVec,center,radius);
			}
		}

		
		particleCount = particlesVec.size();
		HANDLE_ERROR(hipMalloc(&particles, particleCount * sizeof(Particle)));
		HANDLE_ERROR(hipMalloc(&particlesCopy, particleCount * sizeof(Particle)));

		HANDLE_ERROR(hipMemcpy(particles, particlesVec.data(), particleCount * sizeof(Particle), hipMemcpyHostToDevice));

		numThreads = min(1024, particleCount);
		numBlocks = divUp(particleCount, numThreads);

		gridSize.x = ceil(gridPhysicalSize.x / kernelRadius);
		gridSize.y = ceil(gridPhysicalSize.y / kernelRadius);
		gridSize.z = ceil(gridPhysicalSize.z / kernelRadius);

		cellCount = gridSize.x * gridSize.y * gridSize.z;

		HANDLE_ERROR(hipMalloc(&particleIndices, particleCount * sizeof(*particleIndices)));

		HANDLE_ERROR(hipMalloc(&particleHashes, particleCount * sizeof(*particleHashes)));
		HANDLE_ERROR(hipMalloc(&cellBegin, cellCount * sizeof(*cellBegin)));
		HANDLE_ERROR(hipMalloc(&cellEnd, cellCount * sizeof(*cellEnd)));



		pointSprites = std::make_shared<PointSprites>(particleCount);


		computeRestDensity();
		HANDLE_ERROR(hipMemcpy(particlesVec.data(), particles, particleCount * sizeof(Particle), hipMemcpyDeviceToHost));

		float totalRestDensity = 0;
		float maxDensity = 0;
		float minDensity = 99999;
		for (Particle& p : particlesVec) {
			totalRestDensity += p.density;
			maxDensity = max(maxDensity, p.density);
			minDensity = min(minDensity, p.density);
		}
		restDensity = totalRestDensity / (float)particleCount;


		float variance = 0;
		for (Particle& p : particlesVec) {
			variance += pow(p.density - restDensity, 2);
		}
		variance /= (float)particleCount;

		std::cout << "particle count : " << particleCount << std::endl;


		std::cout << "spacing : " << particleSpacing << std::endl;
		std::cout << "kernel radius : " << kernelRadius << std::endl;
		std::cout << "rho0 : " << restDensity << std::endl;

		std::cout << "variance : " << variance << std::endl;
		std::cout << "gridSize.x : " << gridSize.x << std::endl;



		mesher = std::make_shared<Mesher>(gridPhysicalSize, particleSpacing, particleCount, numBlocks, numThreads);
		meshRenderer = std::make_shared<FluidMeshRenderer>(mesher->triangleCount);
	}

	void Fluid::computeRestDensity() {
		performSpatialHashing2(particleIndices, particleHashes, particles, particlesCopy, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

		predictDensityAndPressureImpl << <numBlocks, numThreads >> >
			(particles,particleCount, cellBegin, cellEnd, gridSize, kernelRadius, kernelRadius2, kernelRadius6, kernelRadius9, true, timestep / (float)substeps);

	}


	void Fluid::simulateAsParticles() {
		for (int j = 0; j < 1; ++j) {
			float particlesTimestep = 0.5;

			float beforeHashing = glfwGetTime();

			performSpatialHashing(particleHashes, particles, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

			float afterHashing = glfwGetTime();

			integrate << <numBlocks, numThreads >> > (particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, gridPhysicalSize, kernelRadius, particlesTimestep, particleSpacing);

			collide << <numBlocks, numThreads >> > (particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, kernelRadius, particlesTimestep, particleSpacing);
		}
	}

	
	void Fluid::simulationStep() {

#if SIMULATE_PARTICLES_NOT_FLUID
		simulateAsParticles(); return;
#endif

		for (int i = 0; i < substeps; ++i) {

			performSpatialHashing2(particleIndices,particleHashes, particles, particlesCopy, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

			computeExternalForces();
			initPressure();

			int iter = 0;
			while (iter < minIterations || hasBigError()) {
				if (iter > 4) {
					//std::cout << "hit max iters" << std::endl;
					break;
				}
				predictVelocityAndPosition();

				predictDensityAndPressure();

				computePressureForce();

				iter += 1;
			}

			computeNewVelocityAndPosition();
		}
	}

	void Fluid::computeExternalForces() {
		computeExternalForcesImpl << <numBlocks, numThreads >> > (particles, particleCount);
	}

	void Fluid::initPressure() {
		initPressureImpl << <numBlocks, numThreads >> > (particles, particleCount);
	}

	bool Fluid::hasBigError() {
		return true;
	}

	void Fluid::predictVelocityAndPosition() {
		predictVelocityAndPositionImpl << <numBlocks, numThreads >> >
			(particles, particleCount, timestep / (float)substeps, false, particleSpacing, gridPhysicalSize);
	}

	void Fluid::predictDensityAndPressure() {
		predictDensityAndPressureImpl << <numBlocks, numThreads >> >
			(particles, particleCount, cellBegin, cellEnd, gridSize, kernelRadius, kernelRadius2, kernelRadius6, kernelRadius9, false, timestep / (float)substeps);
	}

	void Fluid::computePressureForce() {
		computePressureForceImpl << <numBlocks, numThreads >> >
			(particles,  particleCount, cellBegin, cellEnd, gridSize, kernelRadius, kernelRadius2, kernelRadius6, kernelRadius9 );
	}

	void Fluid::computeNewVelocityAndPosition() {
		predictVelocityAndPositionImpl << <numBlocks, numThreads >> >
			(particles, particleCount, timestep / (float)substeps, true, particleSpacing, gridPhysicalSize);
	}
}