#include "hip/hip_runtime.h"
#include "Fluid_3D_PCISPH.cuh"

#define SIMULATE_PARTICLES_NOT_FLUID 0

namespace Fluid_3D_PCISPH {
	// this is not for PCISPH.
	// It is used for a pure particle simulation, same as the one in CUDA samples
	__global__ void collide(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius, float timestep, float spacing) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / cellSize;
		thisCell.y = pos.y / cellSize;
		thisCell.z = pos.z / cellSize;

		float3 force = { 0,0,0 };

		float collideDist = spacing;;

#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {

						if (j != index) {
							Particle& that = particles[j];
							float3 relPos = that.position - particle.position;
							float dist = length(relPos);

							if (dist < collideDist) {
								float3 norm = relPos / dist;

								// relative velocity
								float3 relVel = that.velosity - particle.velosity;

								// relative tangential velocity
								float3 tanVel = relVel - (dot(relVel, norm) * norm);

								// spring force
								force += -0.5 * (collideDist - dist) * norm;
								// dashpot (damping) force
								force += 0.02 * relVel;
								// tangential shear force
								force += 0.1 * tanVel;
							}

						}

					}
				}
			}
		}

		particle.velosity += force;
	}
	// this is not for PCISPH.
	// It is used for a pure particle simulation, same as the one in CUDA samples
	__global__ void integrate(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float3 gridDimension, float kernelRadius, float timestep, float spacing) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		particle.velosity += make_float3(0, -0.0003, 0) * timestep;

		float3 pos = particle.position;
		float3 vel = particle.velosity;

		pos += timestep * vel;

		float bounce = -0.5;

		if (pos.x < spacing) {
			pos.x = spacing;
			vel.x *= bounce;;
		}

		if (pos.x > gridDimension.x - spacing) {
			pos.x = gridDimension.x - spacing;
			vel.x *= bounce;;
		}

		if (pos.y < spacing) {
			pos.y = spacing;
			vel.y *= bounce;;
		}

		if (pos.y > gridDimension.y - spacing) {
			pos.y = gridDimension.y - spacing;
			vel.y *= bounce;;
		}

		if (pos.z < spacing) {
			pos.z = spacing;
			vel.z *= bounce;;
		}

		if (pos.z > gridDimension.z - spacing) {
			pos.z = gridDimension.z - spacing;
			vel.z *= bounce;;
		}



		particle.position = pos;
		particle.velosity = vel;

	}













	__global__ void computeExternalForcesImpl(Particle* particles, int particleCount) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];
		particle.acceleration = make_float3(0, -9.8, 0);

	}

	__global__ void initPressureImpl(Particle* particles, int particleCount) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];
		particle.pressure = 0;
		particle.pressureForces = make_float3(0, 0, 0);

	}


	__global__ void predictVelocityAndPositionImpl(Particle* particles, int particleCount, float timestep, bool setAsActual, float spacing, float3 gridDimension) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;

		Particle& particle = particles[index];

		float3 acc = particle.acceleration + particle.pressureForces;
		float3 vel = particle.velosity + acc * timestep;
		float3 pos = particle.position + vel * timestep;

		float bounce = -0.0;

		float minDistanceFromWall = spacing / 2;

		if (pos.x < minDistanceFromWall) {
			pos.x = minDistanceFromWall;
			vel.x *= bounce;;
		}

		if (pos.x > gridDimension.x - minDistanceFromWall) {
			pos.x = gridDimension.x - minDistanceFromWall;
			vel.x *= bounce;;
		}

		if (pos.y < minDistanceFromWall) {
			pos.y = minDistanceFromWall;
			vel.y *= bounce;;
		}

		if (pos.y > gridDimension.y - minDistanceFromWall) {
			pos.y = gridDimension.y - minDistanceFromWall;
			vel.y *= bounce;;
		}

		if (pos.z < minDistanceFromWall) {
			pos.z = minDistanceFromWall;
			vel.z *= bounce;;
		}

		if (pos.z > gridDimension.z - minDistanceFromWall) {
			pos.z = gridDimension.z - minDistanceFromWall;
			vel.z *= bounce;;
		}

		if (setAsActual) {
			particle.position = pos;
			particle.velosity = vel;
		}
		else {
			particle.predictedPosition = pos;
			particle.predictedVelocity = vel;
		}

	}

	__global__ void predictDensityAndPressureImpl(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius, bool setAsRest, float timestep) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / cellSize;
		thisCell.y = pos.y / cellSize;
		thisCell.z = pos.z / cellSize;

		float rho0 = particle.restDensity;

		float beta = timestep * timestep * 2 / (rho0 * rho0);

		float density = 0;

		float3 sumGradW = make_float3(0, 0, 0);
		float sumGradWDot = 0;

#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {
						Particle& that = particles[j];
						float3 posDiff = particle.predictedPosition - that.predictedPosition;
						density += poly6(posDiff, kernelRadius);
						float3 gradW = spikey_grad(posDiff, kernelRadius);
						sumGradW += gradW;
						sumGradWDot += dot(gradW, gradW);
					}
				}
			}
		}
		particle.density = density;

		if (setAsRest) {
			particle.restDensity = density;
		}


		float rhoError = density - rho0;
		float correctionCoeff = 1.0 / (beta * (dot(sumGradW, sumGradW) + sumGradWDot));

		correctionCoeff = 50.0;

		float pressureCorrection = correctionCoeff * rhoError;
		particle.pressure += pressureCorrection;

		if (index == 666) {
			//printf("rho: %f \n", density);
			//printf("stiff: %f \n", correctionCoeff);
		}
	}

	__global__ void computePressureForceImpl(Particle* particles, float cellSize, int particleCount, int* cellBegin, int* cellEnd, int3 gridSize, float kernelRadius) {
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= particleCount) return;


		Particle& particle = particles[index];

		float3 pos = particle.position;
		int3 thisCell;

		thisCell.x = pos.x / cellSize;
		thisCell.y = pos.y / cellSize;
		thisCell.z = pos.z / cellSize;

		float3 force = make_float3(0, 0, 0);

#pragma unroll
		for (int dx = -1; dx <= 1; ++dx) {
#pragma unroll
			for (int dy = -1; dy <= 1; ++dy) {
#pragma unroll
				for (int dz = -1; dz <= 1; ++dz) {
					int x = thisCell.x + dx;
					int y = thisCell.y + dy;
					int z = thisCell.z + dz;
					if (x < 0 || x >= gridSize.x || y < 0 || y >= gridSize.y || z < 0 || z >= gridSize.z) {
						continue;
					}
					int hash = x * gridSize.y * gridSize.z + y * gridSize.z + z;
					if (cellBegin[hash] == -1) {
						continue;
					}
					for (int j = cellBegin[hash]; j <= cellEnd[hash]; ++j) {
						Particle that = particles[j];
						force -= spikey_grad(particle.predictedPosition - that.predictedPosition, kernelRadius)
							* ((that.pressure / (that.density * that.density)) + (particle.pressure / (particle.density * particle.density)));
					}
				}
			}
		}
		particle.pressureForces = force;

	}







	Fluid::Fluid() {

	}

	void Fluid::draw(const DrawCommand& drawCommand){
		skybox.draw(drawCommand);
		container.draw(drawCommand);

		if (drawCommand.renderMode == RenderMode::Mesh) {
			mesher->mesh(particles, particlesCopy, particleHashes, particleIndices, meshRenderer->coordsDevice);
			hipDeviceSynchronize();
			meshRenderer->draw(drawCommand, skybox.texSkyBox);
		}
		else {
			updatePositionsVBO << <numBlocks, numThreads >> > (particles, pointSprites->positionsDevice, particleCount);
			pointSprites->draw(drawCommand, particleSpacing/2, skybox.texSkyBox);
		}

	}

	void Fluid::createSquareFluid(std::vector<Particle>& particlesVec, float3 minPos, float3 maxPos) {
		float3 minPhysicalPos = {
			minPos.x * gridDimension.x,
			minPos.y* gridDimension.y,
			minPos.z* gridDimension.z,
		};
		minPhysicalPos += make_float3(1, 1, 1) * particleSpacing*0.5;
		float3 maxPhysicalPos = {
			maxPos.x* gridDimension.x,
			maxPos.y* gridDimension.y,
			maxPos.z* gridDimension.z,
		};
		maxPhysicalPos -= make_float3(1, 1, 1) * particleSpacing*0.5;
		for (float x = minPhysicalPos.x ; x <= maxPhysicalPos.x; x += particleSpacing) {
			for (float y = minPhysicalPos.y; y <= maxPhysicalPos.y ; y += particleSpacing) {
				for (float z = minPhysicalPos.z; z <= maxPhysicalPos.z ; z += particleSpacing) {
					float jitterMagnitude = 0;
					float3 jitter;
					jitter.x = (random0to1() - 0.5);
					jitter.y = (random0to1() - 0.5);
					jitter.z = (random0to1() - 0.5);
					jitter *= jitterMagnitude;
					particlesVec.emplace_back(make_float3(x, y, z) + jitter);

				}
			}
		}
	}
	void Fluid::createSphereFluid(std::vector<Particle>& particlesVec, float3 center, float radius) {

		float3 minPhysicalPos = {
			0,0,0
		};
		minPhysicalPos += make_float3(1, 1, 1) * particleSpacing * 0.5;
		float3 maxPhysicalPos = gridDimension;
		maxPhysicalPos -= make_float3(1, 1, 1) * particleSpacing * 0.5;

		float3 physicalCenter = {
			center.x * gridDimension.x,
			center.y * gridDimension.y,
			center.z * gridDimension.z
		};

		float physicalRadius = radius * gridDimension.y;

		for (float x = minPhysicalPos.x; x < maxPhysicalPos.x; x += particleSpacing) {
			for (float y = minPhysicalPos.y; y < maxPhysicalPos.y; y += particleSpacing) {
				for (float z = minPhysicalPos.z; z < maxPhysicalPos.z; z += particleSpacing) {

					float3 pos = make_float3(x, y, z);
					float3 jitter = make_float3(1, 1, 1);
					jitter.x *= (random0to1() - 0.5)*particleSpacing*0.01;
					jitter.y *= (random0to1() - 0.5) * particleSpacing * 0.01;
					jitter.z *= (random0to1() - 0.5) * particleSpacing * 0.01;

#if  SIMULATE_PARTICLES_NOT_FLUID
					pos += jitter;
#endif //  SIMULATE_PARTICLES_NOT_FLUID


					
					if (length(pos-physicalCenter) < physicalRadius) {
						
						particlesVec.emplace_back(pos);
					}
				}
			}
		}
	}

	void Fluid::init(std::shared_ptr<FluidConfig> config) {

#if SIMULATE_PARTICLES_NOT_FLUID

		kernelRadius = gridDimension.x / 64;
		particleSpacing = kernelRadius / 2;
		
#else
		particleSpacing = pow(gridDimension.x * gridDimension.y * gridDimension.z / particleCountWhenFull, 1.0 / 3.0);
		kernelRadius = particleSpacing * kernelRadiusToSpacingRatio;
#endif

		

		std::vector<Particle> particlesVec;

		std::shared_ptr<FluidConfig3D> config3D = std::static_pointer_cast<FluidConfig3D, FluidConfig>(config);
		for (const InitializationVolume& vol : config3D->initialVolumes) {
			if (vol.shapeType == ShapeType::Square) {
				float3 minPos = make_float3(vol.params[0], vol.params[1], vol.params[2]);
				float3 maxPos = make_float3(vol.params[3], vol.params[4], vol.params[5]);
				createSquareFluid(particlesVec,minPos,maxPos);
			}
			else if (vol.shapeType == ShapeType::Sphere) {
				float3 center = make_float3(vol.params[0], vol.params[1], vol.params[2]);
				float radius = vol.params[3];
				createSphereFluid(particlesVec,center,radius);
			}
		}

		
		particleCount = particlesVec.size();
		HANDLE_ERROR(hipMalloc(&particles, particleCount * sizeof(Particle)));
		HANDLE_ERROR(hipMalloc(&particlesCopy, particleCount * sizeof(Particle)));

		HANDLE_ERROR(hipMemcpy(particles, particlesVec.data(), particleCount * sizeof(Particle), hipMemcpyHostToDevice));

		numThreads = min(1024, particleCount);
		numBlocks = divUp(particleCount, numThreads);

		gridSize.x = ceil(gridDimension.x / kernelRadius);
		gridSize.y = ceil(gridDimension.y / kernelRadius);
		gridSize.z = ceil(gridDimension.z / kernelRadius);

		cellCount = gridSize.x * gridSize.y * gridSize.z;

		HANDLE_ERROR(hipMalloc(&particleIndices, particleCount * sizeof(*particleIndices)));

		HANDLE_ERROR(hipMalloc(&particleHashes, particleCount * sizeof(*particleHashes)));
		HANDLE_ERROR(hipMalloc(&cellBegin, cellCount * sizeof(*cellBegin)));
		HANDLE_ERROR(hipMalloc(&cellEnd, cellCount * sizeof(*cellEnd)));



		pointSprites = new PointSprites(particleCount);

		std::cout << "particle count : " << particleCount << std::endl;

		computeRestDensity();
		HANDLE_ERROR(hipMemcpy(particlesVec.data(), particles, particleCount * sizeof(Particle), hipMemcpyDeviceToHost));

		float totalRestDensity = 0;
		float maxDensity = 0;
		float minDensity = 99999;
		for (Particle& p : particlesVec) {
			totalRestDensity += p.density;
			maxDensity = max(maxDensity, p.density);
			minDensity = min(minDensity, p.density);
		}
		restDensity = totalRestDensity / (float)particleCount;


		float variance = 0;
		for (Particle& p : particlesVec) {
			variance += pow(p.density - restDensity, 2);
		}
		variance /= (float)particleCount;


		std::cout << "spacing : " << particleSpacing << std::endl;
		std::cout << "kernel radius : " << kernelRadius << std::endl;
		std::cout << "rho0 : " << restDensity << std::endl;

		std::cout << "variance : " << variance << std::endl;
		std::cout << "gridSize.x : " << gridSize.x << std::endl;



		mesher = std::make_shared<Mesher>(gridDimension, particleSpacing, particleCount, numBlocks, numThreads);
		meshRenderer = std::make_shared<FluidMeshRenderer>(mesher->triangleCount);
	}

	void Fluid::computeRestDensity() {
		performSpatialHashing2(particleIndices, particleHashes, particles, particlesCopy, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

		predictDensityAndPressureImpl << <numBlocks, numThreads >> >
			(particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, kernelRadius, true, timestep / (float)substeps);

	}


	void Fluid::simulateAsParticles() {
		for (int j = 0; j < 1; ++j) {
			float particlesTimestep = 0.5;

			float beforeHashing = glfwGetTime();

			performSpatialHashing(particleHashes, particles, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

			float afterHashing = glfwGetTime();

			integrate << <numBlocks, numThreads >> > (particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, gridDimension, kernelRadius, particlesTimestep, particleSpacing);

			collide << <numBlocks, numThreads >> > (particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, kernelRadius, particlesTimestep, particleSpacing);
		}
	}

	
	void Fluid::simulationStep() {

#if SIMULATE_PARTICLES_NOT_FLUID
		simulateAsParticles(); return;
#endif

		for (int i = 0; i < substeps; ++i) {

			performSpatialHashing(particleHashes, particles, particleCount, kernelRadius, gridSize.x, gridSize.y, gridSize.z, numBlocks, numThreads, cellBegin, cellEnd, cellCount);

			computeExternalForces();
			initPressure();

			int iter = 0;
			while (iter < minIterations || hasBigError()) {
				if (iter > 4) {
					//std::cout << "hit max iters" << std::endl;
					break;
				}
				predictVelocityAndPosition();

				predictDensityAndPressure();

				computePressureForce();

				iter += 1;
			}

			computeNewVelocityAndPosition();
		}
	}

	void Fluid::computeExternalForces() {
		computeExternalForcesImpl << <numBlocks, numThreads >> > (particles, particleCount);
	}

	void Fluid::initPressure() {
		initPressureImpl << <numBlocks, numThreads >> > (particles, particleCount);
	}

	bool Fluid::hasBigError() {
		return true;
	}

	void Fluid::predictVelocityAndPosition() {
		predictVelocityAndPositionImpl << <numBlocks, numThreads >> >
			(particles, particleCount, timestep / (float)substeps, false, particleSpacing, gridDimension);
	}

	void Fluid::predictDensityAndPressure() {
		predictDensityAndPressureImpl << <numBlocks, numThreads >> >
			(particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, kernelRadius, false, timestep / (float)substeps);
	}

	void Fluid::computePressureForce() {
		computePressureForceImpl << <numBlocks, numThreads >> >
			(particles, kernelRadius, particleCount, cellBegin, cellEnd, gridSize, kernelRadius);
	}

	void Fluid::computeNewVelocityAndPosition() {
		predictVelocityAndPositionImpl << <numBlocks, numThreads >> >
			(particles, particleCount, timestep / (float)substeps, true, particleSpacing, gridDimension);
	}
}