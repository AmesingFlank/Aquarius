#include "hip/hip_runtime.h"
#include "Fluid_3D_common.cuh"

#include "SPD_Solver.h"


void  applyGravity(float timeStep, MAC_Grid_3D& grid, float gravitationalAcceleration) {
	applyGravityImpl << < grid.numBlocksCell, grid.numThreadsCell >> >
		(grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ, timeStep, gravitationalAcceleration);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("apply forces");

}

void  fixBoundary(MAC_Grid_3D& grid) {
	int sizeX = grid.sizeX;
	int sizeY = grid.sizeY;
	int sizeZ = grid.sizeZ;

	int total, numThreads, numBlocks;

	total = sizeY * sizeZ;
	numThreads = min(1024, total);
	numBlocks = divUp(total, numThreads);
	fixBoundaryX << < numBlocks, numThreads >> > (grid.cells, sizeX, sizeY, sizeZ);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("fix boundary x");

	total = sizeX * sizeZ;
	numThreads = min(1024, total);
	numBlocks = divUp(total, numThreads);
	fixBoundaryY << < numBlocks, numThreads >> > (grid.cells, sizeX, sizeY, sizeZ);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("fix boundary y");

	total = sizeX * sizeY;
	numThreads = min(1024, total);
	numBlocks = divUp(total, numThreads);
	fixBoundaryZ << < numBlocks, numThreads >> > (grid.cells, sizeX, sizeY, sizeZ);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("fix boundary y");

}

void  computeDivergence(MAC_Grid_3D& grid, float restParticlesPerCell) {
	computeDivergenceImpl << < grid.numBlocksCell, grid.numThreadsCell >> >
		(grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ, grid.cellPhysicalSize, restParticlesPerCell);
}

void  solvePressureJacobi(float timeStep, MAC_Grid_3D& grid, int iterations) {

	resetPressureImpl << < grid.numBlocksCell, grid.numThreadsCell >> > (grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ);

	float dt_div_rho_div_dx = 1;

	for (int i = 0; i < iterations; ++i) {
		jacobiImpl << < grid.numBlocksCell, grid.numThreadsCell >> >
			(grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ, dt_div_rho_div_dx, grid.cellPhysicalSize);
	}

}


void  solvePressure(float timeStep, MAC_Grid_3D& grid) {

	int sizeX = grid.sizeX;
	int sizeY = grid.sizeY;
	int sizeZ = grid.sizeZ;
	int numBlocksCell = grid.numBlocksCell;
	int numThreadsCell = grid.numThreadsCell;

	PressureEquation3D* equations = new PressureEquation3D[grid.fluidCount];
	int nnz = 0;
	bool hasNonZeroRHS = false;
	float dt_div_rho_div_dx = 1;


	PressureEquation3D* equationsDevice;
	HANDLE_ERROR(hipMalloc(&equationsDevice, grid.fluidCount * sizeof(PressureEquation3D)));

	bool* hasNonZeroRHS_Device;
	HANDLE_ERROR(hipMalloc(&hasNonZeroRHS_Device, sizeof(*hasNonZeroRHS_Device)));
	HANDLE_ERROR(hipMemset(hasNonZeroRHS_Device, 0, sizeof(*hasNonZeroRHS_Device)));

	constructPressureEquations << < numBlocksCell, numThreadsCell >> >
		(grid.cells, sizeX, sizeY, sizeZ, equationsDevice, dt_div_rho_div_dx, hasNonZeroRHS_Device);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("construct eqns");


	HANDLE_ERROR(hipMemcpy(equations, equationsDevice, grid.fluidCount * sizeof(PressureEquation3D),
		hipMemcpyDeviceToHost));
	HANDLE_ERROR(
		hipMemcpy(&hasNonZeroRHS, hasNonZeroRHS_Device, sizeof(hasNonZeroRHS), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(equationsDevice));
	HANDLE_ERROR(hipFree(hasNonZeroRHS_Device));

	hipDeviceSynchronize();

	for (int i = 0; i < grid.fluidCount; ++i) {
		nnz += equations[i].termCount;
	}

	//std::cout<<"nnz is "<<nnz<<std::endl;


	if (!hasNonZeroRHS) {
		std::cout << "zero RHS" << std::endl;
		return;
	}


	//number of rows == number of variables == number of fluid cells
	int numVariables = grid.fluidCount;



	//construct the matrix of the linear equations
	int nnz_A = nnz;
	double* A_host = (double*)malloc(nnz_A * sizeof(*A_host));
	int* A_rowPtr_host = (int*)malloc((numVariables + 1) * sizeof(*A_rowPtr_host));
	int* A_colInd_host = (int*)malloc(nnz_A * sizeof(*A_colInd_host));

	//construct a symmetric copy, used for computing the preconditioner
	int nnz_R = (nnz - numVariables) / 2 + numVariables;
	nnz_R = numVariables;
	double* R_host = (double*)malloc(nnz_R * sizeof(*R_host));
	int* R_rowPtr_host = (int*)malloc((numVariables + 1) * sizeof(*R_rowPtr_host));
	int* R_colInd_host = (int*)malloc(nnz_R * sizeof(*R_colInd_host));

	for (int row = 0, i = 0; row < numVariables; ++row) {
		PressureEquation3D& thisEquation = equations[row];
		A_rowPtr_host[row] = i;

		for (int term = 0; term < thisEquation.termCount; ++term) {
			//if(thisEquation.termsIndex[term] > row) continue;
			A_host[i] = thisEquation.termsCoeff[term];
			A_colInd_host[i] = thisEquation.termsIndex[term];
			++i;
		}

	}

	for (int row = 0, i = 0; row < numVariables; ++row) {
		PressureEquation3D& thisEquation = equations[row];
		R_rowPtr_host[row] = i;
		for (int term = 0; term < thisEquation.termCount; ++term) {
			if (thisEquation.termsIndex[term] < row) continue;
			R_host[i] = thisEquation.termsCoeff[term];
			R_host[i] = 1;
			if (thisEquation.termsIndex[term] != row) continue;
			R_colInd_host[i] = thisEquation.termsIndex[term];
			++i;
		}
	}

	A_rowPtr_host[numVariables] = nnz_A;
	R_rowPtr_host[numVariables] = nnz_R;

	double* A_device;
	HANDLE_ERROR(hipMalloc(&A_device, nnz_A * sizeof(*A_device)));
	HANDLE_ERROR(hipMemcpy(A_device, A_host, nnz_A * sizeof(*A_device), hipMemcpyHostToDevice));

	int* A_rowPtr_device;
	HANDLE_ERROR(hipMalloc(&A_rowPtr_device, (numVariables + 1) * sizeof(*A_rowPtr_device)));
	HANDLE_ERROR(hipMemcpy(A_rowPtr_device, A_rowPtr_host, (numVariables + 1) * sizeof(*A_rowPtr_device),
		hipMemcpyHostToDevice));

	int* A_colInd_device;
	HANDLE_ERROR(hipMalloc(&A_colInd_device, nnz_A * sizeof(*A_colInd_device)));
	HANDLE_ERROR(hipMemcpy(A_colInd_device, A_colInd_host, nnz_A * sizeof(*A_colInd_device),
		hipMemcpyHostToDevice));

	hipsparseMatDescr_t descrA;
	HANDLE_ERROR(hipsparseCreateMatDescr(&descrA));
	//hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
	//hipsparseSetMatFillMode(descrA, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

	SparseMatrixCSR A(numVariables, numVariables, A_device, A_rowPtr_device, A_colInd_device, descrA, nnz_A);

	double* R_device;
	HANDLE_ERROR(hipMalloc(&R_device, nnz_R * sizeof(*R_device)));
	HANDLE_ERROR(hipMemcpy(R_device, R_host, nnz_R * sizeof(*R_device), hipMemcpyHostToDevice));

	int* R_rowPtr_device;
	HANDLE_ERROR(hipMalloc(&R_rowPtr_device, (numVariables + 1) * sizeof(*R_rowPtr_device)));
	HANDLE_ERROR(hipMemcpy(R_rowPtr_device, R_rowPtr_host, (numVariables + 1) * sizeof(*R_rowPtr_device),
		hipMemcpyHostToDevice));

	int* R_colInd_device;
	HANDLE_ERROR(hipMalloc(&R_colInd_device, nnz_R * sizeof(*R_colInd_device)));
	HANDLE_ERROR(hipMemcpy(R_colInd_device, R_colInd_host, nnz_R * sizeof(*R_colInd_device),
		hipMemcpyHostToDevice));

	hipsparseMatDescr_t descrR;
	HANDLE_ERROR(hipsparseCreateMatDescr(&descrR));
	hipsparseSetMatType(descrR, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
	hipsparseSetMatFillMode(descrR, HIPSPARSE_FILL_MODE_UPPER);
	//hipsparseSetMatType(descrR, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatDiagType(descrR, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descrR, HIPSPARSE_INDEX_BASE_ZERO);

	SparseMatrixCSR R(numVariables, numVariables, R_device, R_rowPtr_device, R_colInd_device, descrR, nnz_R);
	/*
			cusparseSolveAnalysisInfo_t ic0Info = 0;

			HANDLE_ERROR(cusparseCreateSolveAnalysisInfo(&ic0Info));

			HANDLE_ERROR(cusparseDcsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
									numVariables,nnz_R, descrR, R_device, R_rowPtr_device, R_colInd_device, ic0Info));

			HANDLE_ERROR(cusparseDcsric0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, numVariables, descrR,
											 R_device, R_rowPtr_device, R_colInd_device, ic0Info));

			HANDLE_ERROR(cusparseDestroySolveAnalysisInfo(ic0Info));
	*/
	hipsparseSetMatType(descrR, HIPSPARSE_MATRIX_TYPE_TRIANGULAR);

	//RHS vector
	double* f_host = (double*)malloc(numVariables * sizeof(*f_host));
	for (int i = 0; i < numVariables; ++i) {
		f_host[i] = equations[i].RHS;
	}

	//solve the pressure equation
	double* result_device = solveSPD2(A, R, f_host, numVariables);

	double* result_host = new double[numVariables];
	HANDLE_ERROR(hipMemcpy(result_host, result_device, numVariables * sizeof(*result_host),
		hipMemcpyDeviceToHost));


	setPressure << < numBlocksCell, numThreadsCell >> > (grid.cells, sizeX, sizeY, sizeZ, result_device);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("set pressure");



	A.free();
	R.free();
	free(f_host);
	HANDLE_ERROR(hipFree(result_device));
	delete[](result_host);

	delete[] equations;

}

void  updateVelocityWithPressure(float timeStep, MAC_Grid_3D& grid) {
	float dt_div_rho_div_dx = 1;
	updateVelocityWithPressureImpl << < grid.numBlocksCell, grid.numThreadsCell >> > (grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ, dt_div_rho_div_dx);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR("update velocity with pressure");
}


void  extrapolateVelocity(float timeStep, MAC_Grid_3D& grid) {

	//used to decide how far to extrapolate
	float maxSpeed = grid.getMaxSpeed();

	float maxDist = ceil((maxSpeed * timeStep) / grid.cellPhysicalSize);
	//maxDist=4;
	//std::cout<<"maxDist "<<maxDist<<std::endl;

	for (int distance = 0; distance < maxDist; ++distance) {
		extrapolateVelocityByOne << < grid.numBlocksCell, grid.numThreadsCell >> > (grid.cells, grid.sizeX, grid.sizeY, grid.sizeZ);
		hipDeviceSynchronize();
		CHECK_CUDA_ERROR("extrapolate vel");
	}
}