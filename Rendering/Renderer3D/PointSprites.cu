#include "PointSprites.h"
#include "../DrawCommand.h"

void PointSprites::draw(const DrawCommand& drawCommand, float radius, int skybox) {
	//return;
	if (drawCommand.renderMode == RenderMode::Particles) {
		drawSimple(drawCommand, radius); return;
	}
	
	GLuint normalTexture = screenSpaceNormal.generateNormalTexture([&]() 
		{
			drawDepth(drawCommand,radius);
		},
		6, 5, 6, 0.1, drawCommand);
	GLuint depthTexture = screenSpaceNormal.lastDepthTexture;

	drawThickness(drawCommand, radius);

	drawScreen(drawCommand, skybox,normalTexture,depthTexture,radius);
	printGLError();
}

void PointSprites::initRenderer() {

	pointsVBO_host = new float[count * stride];

	simpleShader = std::make_shared<Shader>(
		Shader::SHADERS_PATH("PointSprites_points_vs.glsl"), 
		Shader::SHADERS_PATH("PointSprites_simple_fs.glsl")
	);

	phaseThicknessShader = std::make_shared<Shader>(
		Shader::SHADERS_PATH("PointSprites_points_vs.glsl"), 
		Shader::SHADERS_PATH("PointSprites_phase_fs.glsl")
	);

	depthShader = std::make_shared<Shader>(
		Shader::SHADERS_PATH("PointSprites_points_vs.glsl"),
		Shader::SHADERS_PATH("PointSprites_depth_fs.glsl")
	);
	screenShader = std::make_shared<Shader>(
		Shader::SHADERS_PATH("PointSprites_screen_vs.glsl"),
		Shader::SHADERS_PATH("PointSprites_screen_fs.glsl")
	);


	thicknessShader = std::make_shared<Shader>(
		Shader::SHADERS_PATH("PointSprites_points_vs.glsl"),
		Shader::SHADERS_PATH("PointSprites_thickness_fs.glsl")
	);

	// used by multiple shaders. location specified as common value in all shader code
	GLint pointsPositionLocation = glGetAttribLocation(simpleShader->program, "position");
	GLint pointsVolumeFractionsLocation = glGetAttribLocation(phaseThicknessShader->program, "volumeFractions");



	glGenVertexArrays(1, &pointsVAO);
	glGenBuffers(1, &pointsVBO);
	glBindVertexArray(pointsVAO);
	glBindBuffer(GL_ARRAY_BUFFER, pointsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float) * count * 7, pointsVBO_host, GL_STATIC_DRAW);

	glEnableVertexAttribArray(pointsPositionLocation);
	glVertexAttribPointer(pointsPositionLocation, 3, GL_FLOAT, GL_FALSE, sizeof(float) * stride, 0);

	glEnableVertexAttribArray(pointsVolumeFractionsLocation);
	glVertexAttribPointer(pointsVolumeFractionsLocation, 4, GL_FLOAT, GL_FALSE, sizeof(float) * stride, (void*)(sizeof(float) * 3));

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaResourceVBO, pointsVBO, cudaGraphicsMapFlagsNone));

	size_t  size;
	HANDLE_ERROR(hipGraphicsMapResources(1, &cudaResourceVBO, NULL));
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&positionsDevice, &size, cudaResourceVBO));

	glBindVertexArray(0);





	glGenTextures(1, &depthTextureNDC);
	glBindTexture(GL_TEXTURE_2D, depthTextureNDC);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_DEPTH_COMPONENT, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_DEPTH_COMPONENT, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &thicknessTexture);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RGBA, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &phaseThicknessTexture);
	glBindTexture(GL_TEXTURE_2D, phaseThicknessTexture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RGBA, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenFramebuffers(1, &FBO);
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_2D, depthTextureNDC, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT3, GL_TEXTURE_2D, thicknessTexture, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT4, GL_TEXTURE_2D, phaseThicknessTexture,0);


	checkFramebufferComplete();
	glBindFramebuffer(GL_FRAMEBUFFER, 0);


	

	glGenVertexArrays(1, &quadVAO);
	glGenBuffers(1, &quadVBO);
	glBindVertexArray(quadVAO);
	glBindBuffer(GL_ARRAY_BUFFER, quadVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), quadVertices, GL_STATIC_DRAW);

	GLuint quad_vPos_location, quad_texCoord_location;
	quad_vPos_location = glGetAttribLocation(screenShader->program, "vPos");
	quad_texCoord_location = glGetAttribLocation(screenShader->program, "texCoord");

	glEnableVertexAttribArray(quad_vPos_location);
	glVertexAttribPointer(quad_vPos_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(quad_texCoord_location);
	glVertexAttribPointer(quad_texCoord_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(2 * sizeof(float)));
	printGLError();

	delete[] pointsVBO_host;

}



void PointSprites::drawDepth(const DrawCommand& drawCommand, float radius) {
	
	depthShader->use();

	prepareShader(depthShader,drawCommand,radius);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

}


void PointSprites::drawThickness(const DrawCommand& drawCommand, float radius) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glEnable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_ONE, GL_ONE, GL_ONE, GL_ONE);

	glClear(GL_DEPTH_BUFFER_BIT);



	thicknessShader->use();

	prepareShader(thicknessShader,drawCommand,radius);

	GLenum bufs[] = { GL_COLOR_ATTACHMENT3 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0, 0, 0, 0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glEnable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	//glBlendFuncSeparate(GL_ONE, GL_ZERO, GL_ONE, GL_ZERO);

}

void PointSprites::drawScreen(const DrawCommand& drawCommand, int skybox,GLuint normalTexture,GLuint depthTexture,float radius) {
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);

	screenShader->use();
	glBindVertexArray(quadVAO);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, depthTexture);
	screenShader->setUniform1i("depthTexture", 0);

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, normalTexture);
	screenShader->setUniform1i("normalTexture", 1);

	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	screenShader->setUniform1i("thicknessTexture", 2);

	glActiveTexture(GL_TEXTURE3);
	glBindTexture(GL_TEXTURE_CUBE_MAP, skybox);
	screenShader->setUniform1i("skybox", 3);

	prepareShader(screenShader,drawCommand,radius);


	glm::mat4 inverseView = glm::inverse(drawCommand.view);

	screenShader->setUniformMat4("inverseView",inverseView);


	screenShader->setUniform1f("zoom", drawCommand.zoom);

	glDrawArrays(GL_TRIANGLES, 0, 6);

}

PointSprites::PointSprites(int count_) :count(count_) {

	

	initRenderer();

};




void PointSprites::drawSimple(const DrawCommand& drawCommand, float radius) {

	glEnable(GL_BLEND);
	//glDisable(GL_DEPTH_TEST);
	glEnable(GL_DEPTH_TEST);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glBlendEquation(GL_FUNC_ADD);


	simpleShader->use();
	prepareShader(simpleShader,drawCommand,radius);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);
	//glPointSize(50);
	glDrawArrays(GL_POINTS, 0, count);
	glEnable(GL_DEPTH_TEST);


}


void PointSprites::drawPhaseThickness(const DrawCommand& drawCommand, float radius) {



	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glEnable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_ONE, GL_ONE, GL_ONE, GL_ONE);

	glClear(GL_DEPTH_BUFFER_BIT);

	phaseThicknessShader->use();
	prepareShader(phaseThicknessShader,drawCommand,radius);


	GLenum bufs[] = { GL_COLOR_ATTACHMENT4 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0, 0, 0, 0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glEnable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);


}


void PointSprites::prepareShader(std::shared_ptr<Shader> shader, const DrawCommand& drawCommand, float radius) {
	shader->use();

	shader->setUniform1f("windowWidth", drawCommand.windowWidth);

	shader->setUniform1f("windowHeight", drawCommand.windowHeight);

	shader->setUniform1f("radius", radius);

	shader->setUniform3f("cameraPosition", drawCommand.cameraPosition);


	shader->setUniformMat4("model", model);
	shader->setUniformMat4("view", drawCommand.view);
	shader->setUniformMat4("projection", drawCommand.projection);

}

PointSprites::~PointSprites() {
	HANDLE_ERROR(hipGraphicsUnmapResources(1,&cudaResourceVBO));
	HANDLE_ERROR(hipGraphicsUnregisterResource(cudaResourceVBO));
	glDeleteBuffers(1, &pointsVBO);
	glDeleteVertexArrays(1, &pointsVAO);

	glDeleteBuffers(1, &quadVBO);
	glDeleteVertexArrays(1, &quadVAO);

	glDeleteTextures(1, &depthTextureNDC);
	glDeleteTextures(1, &thicknessTexture);
	glDeleteTextures(1, &phaseThicknessTexture);

	glDeleteFramebuffers(1, &FBO);
}