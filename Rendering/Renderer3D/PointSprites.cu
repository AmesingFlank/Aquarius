#include "PointSprites.h"
#include "../DrawCommand.h"

void PointSprites::draw(const DrawCommand& drawCommand, float radius, int skybox) {
	//drawSimple(drawCommand, radius); return;

	renderDepth(drawCommand, radius);

	smoothDepth(drawCommand, 6, 5, 6, 0.1);

	renderNormal(drawCommand);

	renderThickness(drawCommand, radius);


	renderFinal(drawCommand, skybox);
	printGLError();
}

void PointSprites::initScreenSpaceRenderer() {


	glGenTextures(1, &depthTextureNDC);
	glBindTexture(GL_TEXTURE_2D, depthTextureNDC);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_DEPTH_COMPONENT, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_DEPTH_COMPONENT, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &depthTextureA);
	glBindTexture(GL_TEXTURE_2D, depthTextureA);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RED, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &depthTextureB);
	glBindTexture(GL_TEXTURE_2D, depthTextureB);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RED, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &normalTexture);
	glBindTexture(GL_TEXTURE_2D, normalTexture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RGBA, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &thicknessTexture);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RGBA, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenFramebuffers(1, &FBO);
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_2D, depthTextureNDC, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, depthTextureA, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT1, GL_TEXTURE_2D, depthTextureB, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT2, GL_TEXTURE_2D, normalTexture, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT3, GL_TEXTURE_2D, thicknessTexture, 0);


	checkFramebufferComplete();
	glBindFramebuffer(GL_FRAMEBUFFER, 0);


	depthShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_depth_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_depth_fs.glsl").c_str()
	);
	renderShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_render_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_render_fs.glsl").c_str()
	);
	normalShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_normal_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_normal_fs.glsl").c_str()
	);
	thicknessShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_thickness_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_thickness_fs.glsl").c_str()
	);
	smoothShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_smooth_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_smooth_fs.glsl").c_str()
	);


	glGenVertexArrays(1, &quadVAO);
	glGenBuffers(1, &quadVBO);
	glBindVertexArray(quadVAO);
	glBindBuffer(GL_ARRAY_BUFFER, quadVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), quadVertices, GL_STATIC_DRAW);

	GLuint quad_vPos_location, quad_texCoord_location;
	quad_vPos_location = glGetAttribLocation(renderShader->Program, "vPos");
	quad_texCoord_location = glGetAttribLocation(renderShader->Program, "texCoord");

	glEnableVertexAttribArray(quad_vPos_location);
	glVertexAttribPointer(quad_vPos_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(quad_texCoord_location);
	glVertexAttribPointer(quad_texCoord_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(2 * sizeof(float)));
	printGLError();

}



void PointSprites::renderDepth(const DrawCommand& drawCommand, float radius) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glDisable(GL_BLEND);

	glClear(GL_DEPTH_BUFFER_BIT);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	depthShader->Use();

	GLenum bufs[] = { GL_COLOR_ATTACHMENT0 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0, 0, 0, 0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "model")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "view")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "projection")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(depthShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(depthShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(depthShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(depthShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_BLEND);

	lastDepthTexture = depthTextureA;

}

void PointSprites::smoothDepth(const DrawCommand& drawCommand, int iterations, int smoothRadius, float sigma_d, float sigma_r) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glDisable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	smoothShader->Use();
	glBindVertexArray(quadVAO);

	GLuint windowWidthLocation = glGetUniformLocation(smoothShader->Program, "windowWidth");
	glUniform1f(windowWidthLocation, drawCommand.windowWidth);

	GLuint windowHeightLocation = glGetUniformLocation(smoothShader->Program, "windowHeight");
	glUniform1f(windowHeightLocation, drawCommand.windowHeight);

	GLuint smoothRadiusXLocation = glGetUniformLocation(smoothShader->Program, "smoothRadiusX");
	GLuint smoothRadiusYLocation = glGetUniformLocation(smoothShader->Program, "smoothRadiusY");


	GLuint sigma_d_location = glGetUniformLocation(smoothShader->Program, "sigma_d");
	glUniform1f(sigma_d_location, sigma_d);

	GLuint sigma_r_location = glGetUniformLocation(smoothShader->Program, "sigma_r");
	glUniform1f(sigma_r_location, sigma_r);

	int smoothRadiusX = smoothRadius;
	int smoothRadiusY = smoothRadius;


	for (int i = 0; i < iterations; i++) {

		GLuint targetAttachment;
		GLuint nextDepthTexture;

		if (lastDepthTexture == depthTextureA) {
			targetAttachment = GL_COLOR_ATTACHMENT1;
			nextDepthTexture = depthTextureB;
		}
		else {
			targetAttachment = GL_COLOR_ATTACHMENT0;
			nextDepthTexture = depthTextureA;
		}

		glBindFramebuffer(GL_FRAMEBUFFER, FBO);

		smoothShader->Use();
		glBindVertexArray(quadVAO);



		GLenum bufs[] = { targetAttachment };
		glDrawBuffers(1, bufs);


		static const float zero[] = { 0,0,0,0 };
		glClearBufferfv(GL_COLOR, 0, zero);


		glUniform1i(smoothRadiusXLocation, smoothRadiusX);
		glUniform1i(smoothRadiusYLocation, smoothRadiusY);

		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, lastDepthTexture);
		GLuint depthTextureLocation = glGetUniformLocation(smoothShader->Program, "depthTexture");
		glUniform1i(depthTextureLocation, 0);

		glDrawArrays(GL_TRIANGLES, 0, 6);

		glBindFramebuffer(GL_FRAMEBUFFER, 0);
		lastDepthTexture = nextDepthTexture;
		std::swap(smoothRadiusX, smoothRadiusY);
	}


	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_BLEND);
	glEnable(GL_DEPTH_TEST);
}

void PointSprites::renderNormal(const DrawCommand& drawCommand) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glDisable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	GLenum bufs[] = { GL_COLOR_ATTACHMENT2 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0,0,0,0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	normalShader->Use();
	glBindVertexArray(quadVAO);


	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, lastDepthTexture);
	GLuint depthTextureLocation = glGetUniformLocation(normalShader->Program, "depthTexture");
	glUniform1i(depthTextureLocation, 0);

	GLuint windowWidthLocation = glGetUniformLocation(normalShader->Program, "windowWidth");
	glUniform1f(windowWidthLocation, drawCommand.windowWidth);

	GLuint windowHeightLocation = glGetUniformLocation(normalShader->Program, "windowHeight");
	glUniform1f(windowHeightLocation, drawCommand.windowHeight);

	GLuint zoomLocation = glGetUniformLocation(normalShader->Program, "zoom");
	glUniform1f(zoomLocation, drawCommand.zoom);

	glDrawArrays(GL_TRIANGLES, 0, 6);


	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_BLEND);
	glEnable(GL_DEPTH_TEST);

}

void PointSprites::renderThickness(const DrawCommand& drawCommand, float radius) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glEnable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_ONE, GL_ONE, GL_ONE, GL_ONE);

	glClear(GL_DEPTH_BUFFER_BIT);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	thicknessShader->Use();

	GLenum bufs[] = { GL_COLOR_ATTACHMENT3 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0, 0, 0, 0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "model")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "view")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "projection")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(thicknessShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(thicknessShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(thicknessShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(thicknessShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glEnable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	//glBlendFuncSeparate(GL_ONE, GL_ZERO, GL_ONE, GL_ZERO);

}

void PointSprites::renderFinal(const DrawCommand& drawCommand, int skybox) {
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);

	renderShader->Use();
	glBindVertexArray(quadVAO);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, lastDepthTexture);
	GLuint depthTextureLocation = glGetUniformLocation(renderShader->Program, "depthTexture");
	glUniform1i(depthTextureLocation, 0);

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, normalTexture);
	GLuint normalTextureLocation = glGetUniformLocation(renderShader->Program, "normalTexture");
	glUniform1i(normalTextureLocation, 1);

	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	GLuint thicknessTextureLocation = glGetUniformLocation(renderShader->Program, "thicknessTexture");
	glUniform1i(thicknessTextureLocation, 2);

	glActiveTexture(GL_TEXTURE3);
	glBindTexture(GL_TEXTURE_CUBE_MAP, skybox);
	GLuint skyboxLocation = glGetUniformLocation(renderShader->Program, "skybox");
	glUniform1i(skyboxLocation, 3);


	GLuint projectionLocation = glGetUniformLocation(renderShader->Program, "projection");
	glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(drawCommand.projection));

	glm::mat4 inverseView = glm::inverse(drawCommand.view);
	GLuint inverseViewLocation = glGetUniformLocation(renderShader->Program, "inverseView");
	glUniformMatrix4fv(inverseViewLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(inverseView));

	GLuint windowWidthLocation = glGetUniformLocation(renderShader->Program, "windowWidth");
	glUniform1f(windowWidthLocation, drawCommand.windowWidth);

	GLuint windowHeightLocation = glGetUniformLocation(renderShader->Program, "windowHeight");
	glUniform1f(windowHeightLocation, drawCommand.windowHeight);

	GLuint zoomLocation = glGetUniformLocation(renderShader->Program, "zoom");
	glUniform1f(zoomLocation, drawCommand.zoom);


	glm::vec3 cameraPos = drawCommand.cameraPosition;
	GLuint cameraPositionLocation = glGetUniformLocation(renderShader->Program, "cameraPosition");
	glUniform3f(cameraPositionLocation, cameraPos.x, cameraPos.y, cameraPos.z);

	glDrawArrays(GL_TRIANGLES, 0, 6);

}

PointSprites::PointSprites(int count_) :count(count_) {

	positionsHost = new float[count * 3];

	basicShader = new Shader(Shader::SHADERS_PATH("PointSprites_vs.glsl").c_str(), Shader::SHADERS_PATH("PointSprites_fs.glsl").c_str(), nullptr);

	points_vPos_location = glGetAttribLocation(basicShader->Program, "position");



	glGenVertexArrays(1, &pointsVAO);
	glGenBuffers(1, &pointsVBO);
	glBindVertexArray(pointsVAO);
	glBindBuffer(GL_ARRAY_BUFFER, pointsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float) * count * 3, positionsHost, GL_STATIC_DRAW);
	glEnableVertexAttribArray(points_vPos_location);
	glVertexAttribPointer(points_vPos_location, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, 0);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaResourceVBO, pointsVBO, cudaGraphicsMapFlagsNone));

	size_t  size;
	HANDLE_ERROR(hipGraphicsMapResources(1, &cudaResourceVBO, NULL));
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&positionsDevice, &size, cudaResourceVBO));

	glBindVertexArray(0);

	initScreenSpaceRenderer();

};




void PointSprites::drawSimple(const DrawCommand& drawCommand, float radius) {

	glEnable(GL_BLEND);
	//glDisable(GL_DEPTH_TEST);
	glEnable(GL_DEPTH_TEST);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glBlendEquation(GL_FUNC_ADD);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	basicShader->Use();
	GLuint modelLocation = glGetUniformLocation(basicShader->Program, "model");
	GLuint viewLocation = glGetUniformLocation(basicShader->Program, "view");
	GLuint projectionLocation = glGetUniformLocation(basicShader->Program, "projection");

	glUniformMatrix4fv(modelLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(viewLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(basicShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(basicShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(basicShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(basicShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);


	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);
	//glPointSize(50);
	glDrawArrays(GL_POINTS, 0, count);
	glEnable(GL_DEPTH_TEST);


}