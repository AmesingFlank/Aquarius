#include "PointSprites.h"
#include "../DrawCommand.h"

void PointSprites::draw(const DrawCommand& drawCommand, float radius, int skybox) {
	//return;
	if (drawCommand.renderMode == RenderMode::Particles) {
		drawSimple(drawCommand, radius); return;
	}
	
	GLuint normalTexture = screenSpaceNormal.generateNormalTexture([&]() 
		{
			renderDepth(drawCommand,radius);
		},
		6, 5, 6, 0.1, drawCommand);
	GLuint depthTexture = screenSpaceNormal.lastDepthTexture;

	renderThickness(drawCommand, radius);

	renderFinal(drawCommand, skybox,normalTexture,depthTexture);
	printGLError();
}

void PointSprites::initRenderer() {

	pointsVBO_host = new float[count * stride];

	pointsShader = new Shader(Shader::SHADERS_PATH("PointSprites_points_vs.glsl").c_str(), Shader::SHADERS_PATH("PointSprites_points_fs.glsl").c_str(), nullptr);

	inkShader = new Shader(Shader::SHADERS_PATH("PointSprites_points_vs.glsl").c_str(), Shader::SHADERS_PATH("PointSprites_ink_fs.glsl").c_str(), nullptr);


	// used by multiple shaders. location specified as common value in all shader code
	GLint pointsPositionLocation = glGetAttribLocation(pointsShader->Program, "position");
	GLint pointsColorLocation = glGetAttribLocation(pointsShader->Program, "color");


	glGenVertexArrays(1, &pointsVAO);
	glGenBuffers(1, &pointsVBO);
	glBindVertexArray(pointsVAO);
	glBindBuffer(GL_ARRAY_BUFFER, pointsVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float) * count * 7, pointsVBO_host, GL_STATIC_DRAW);

	glEnableVertexAttribArray(pointsPositionLocation);
	glVertexAttribPointer(pointsPositionLocation, 3, GL_FLOAT, GL_FALSE, sizeof(float) * stride, 0);

	glEnableVertexAttribArray(pointsColorLocation);
	glVertexAttribPointer(pointsColorLocation, 4, GL_FLOAT, GL_FALSE, sizeof(float) * stride, (void*)(sizeof(float) * 3));

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaResourceVBO, pointsVBO, cudaGraphicsMapFlagsNone));

	size_t  size;
	HANDLE_ERROR(hipGraphicsMapResources(1, &cudaResourceVBO, NULL));
	HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&positionsDevice, &size, cudaResourceVBO));

	glBindVertexArray(0);





	glGenTextures(1, &depthTextureNDC);
	glBindTexture(GL_TEXTURE_2D, depthTextureNDC);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_DEPTH_COMPONENT, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_DEPTH_COMPONENT, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenTextures(1, &thicknessTexture);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WindowInfo::instance().windowWidth, WindowInfo::instance().windowHeight, 0, GL_RGBA, GL_FLOAT, NULL);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glGenFramebuffers(1, &FBO);
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_2D, depthTextureNDC, 0);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT3, GL_TEXTURE_2D, thicknessTexture, 0);


	checkFramebufferComplete();
	glBindFramebuffer(GL_FRAMEBUFFER, 0);


	depthShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_depth_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_depth_fs.glsl").c_str()
	);
	renderShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_render_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_render_fs.glsl").c_str()
	);
	

	thicknessShader = new Shader(
		Shader::SHADERS_PATH("PointSprites_thickness_vs.glsl").c_str(),
		Shader::SHADERS_PATH("PointSprites_thickness_fs.glsl").c_str()
	);

	glGenVertexArrays(1, &quadVAO);
	glGenBuffers(1, &quadVBO);
	glBindVertexArray(quadVAO);
	glBindBuffer(GL_ARRAY_BUFFER, quadVBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), quadVertices, GL_STATIC_DRAW);

	GLuint quad_vPos_location, quad_texCoord_location;
	quad_vPos_location = glGetAttribLocation(renderShader->Program, "vPos");
	quad_texCoord_location = glGetAttribLocation(renderShader->Program, "texCoord");

	glEnableVertexAttribArray(quad_vPos_location);
	glVertexAttribPointer(quad_vPos_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(quad_texCoord_location);
	glVertexAttribPointer(quad_texCoord_location, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(2 * sizeof(float)));
	printGLError();

}



void PointSprites::renderDepth(const DrawCommand& drawCommand, float radius) {
	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	depthShader->Use();


	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "model")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "view")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(glGetUniformLocation(depthShader->Program, "projection")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(depthShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(depthShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(depthShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(depthShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

}


void PointSprites::renderThickness(const DrawCommand& drawCommand, float radius) {
	glBindFramebuffer(GL_FRAMEBUFFER, FBO);
	glEnable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_ONE, GL_ONE, GL_ONE, GL_ONE);

	glClear(GL_DEPTH_BUFFER_BIT);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	thicknessShader->Use();

	GLenum bufs[] = { GL_COLOR_ATTACHMENT3 };
	glDrawBuffers(1, bufs);

	static const float zero[] = { 0, 0, 0, 0 };
	glClearBufferfv(GL_COLOR, 0, zero);

	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "model")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "view")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(glGetUniformLocation(thicknessShader->Program, "projection")
		, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(thicknessShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(thicknessShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(thicknessShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(thicknessShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);

	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);

	glDrawArrays(GL_POINTS, 0, count);

	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glEnable(GL_DEPTH_TEST);

	glBlendFuncSeparate(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA, GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	//glBlendFuncSeparate(GL_ONE, GL_ZERO, GL_ONE, GL_ZERO);

}

void PointSprites::renderFinal(const DrawCommand& drawCommand, int skybox,GLuint normalTexture,GLuint depthTexture) {
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_BLEND);

	renderShader->Use();
	glBindVertexArray(quadVAO);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, depthTexture);
	GLuint depthTextureLocation = glGetUniformLocation(renderShader->Program, "depthTexture");
	glUniform1i(depthTextureLocation, 0);

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, normalTexture);
	GLuint normalTextureLocation = glGetUniformLocation(renderShader->Program, "normalTexture");
	glUniform1i(normalTextureLocation, 1);

	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, thicknessTexture);
	GLuint thicknessTextureLocation = glGetUniformLocation(renderShader->Program, "thicknessTexture");
	glUniform1i(thicknessTextureLocation, 2);

	glActiveTexture(GL_TEXTURE3);
	glBindTexture(GL_TEXTURE_CUBE_MAP, skybox);
	GLuint skyboxLocation = glGetUniformLocation(renderShader->Program, "skybox");
	glUniform1i(skyboxLocation, 3);


	GLuint projectionLocation = glGetUniformLocation(renderShader->Program, "projection");
	glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(drawCommand.projection));

	glm::mat4 inverseView = glm::inverse(drawCommand.view);
	GLuint inverseViewLocation = glGetUniformLocation(renderShader->Program, "inverseView");
	glUniformMatrix4fv(inverseViewLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(inverseView));

	GLuint windowWidthLocation = glGetUniformLocation(renderShader->Program, "windowWidth");
	glUniform1f(windowWidthLocation, drawCommand.windowWidth);

	GLuint windowHeightLocation = glGetUniformLocation(renderShader->Program, "windowHeight");
	glUniform1f(windowHeightLocation, drawCommand.windowHeight);

	GLuint zoomLocation = glGetUniformLocation(renderShader->Program, "zoom");
	glUniform1f(zoomLocation, drawCommand.zoom);


	glm::vec3 cameraPos = drawCommand.cameraPosition;
	GLuint cameraPositionLocation = glGetUniformLocation(renderShader->Program, "cameraPosition");
	glUniform3f(cameraPositionLocation, cameraPos.x, cameraPos.y, cameraPos.z);

	glDrawArrays(GL_TRIANGLES, 0, 6);

}

PointSprites::PointSprites(int count_) :count(count_) {

	

	initRenderer();

};




void PointSprites::drawSimple(const DrawCommand& drawCommand, float radius) {

	glEnable(GL_BLEND);
	//glDisable(GL_DEPTH_TEST);
	glEnable(GL_DEPTH_TEST);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glBlendEquation(GL_FUNC_ADD);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	pointsShader->Use();
	GLuint modelLocation = glGetUniformLocation(pointsShader->Program, "model");
	GLuint viewLocation = glGetUniformLocation(pointsShader->Program, "view");
	GLuint projectionLocation = glGetUniformLocation(pointsShader->Program, "projection");

	glUniformMatrix4fv(modelLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(viewLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(pointsShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(pointsShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(pointsShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(pointsShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);


	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);
	//glPointSize(50);
	glDrawArrays(GL_POINTS, 0, count);
	glEnable(GL_DEPTH_TEST);


}


void PointSprites::drawInk(const DrawCommand& drawCommand, float radius) {

	glEnable(GL_BLEND);
	glDisable(GL_DEPTH_TEST);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	glBlendEquation(GL_FUNC_ADD);

	glm::mat4 view = drawCommand.view;
	glm::mat4 projection = drawCommand.projection;
	glm::vec3 cameraPos = drawCommand.cameraPosition;

	inkShader->Use();
	GLuint modelLocation = glGetUniformLocation(inkShader->Program, "model");
	GLuint viewLocation = glGetUniformLocation(inkShader->Program, "view");
	GLuint projectionLocation = glGetUniformLocation(inkShader->Program, "projection");

	glUniformMatrix4fv(modelLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(model));
	glUniformMatrix4fv(viewLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(view));
	glUniformMatrix4fv(projectionLocation, 1, GL_FALSE, (const GLfloat*)glm::value_ptr(projection));

	glUniform1f(glGetUniformLocation(inkShader->Program, "windowWidth"), drawCommand.windowWidth);
	glUniform1f(glGetUniformLocation(inkShader->Program, "windowHeight"), drawCommand.windowHeight);

	glUniform1f(glGetUniformLocation(inkShader->Program, "radius"), radius);

	glUniform3f(glGetUniformLocation(inkShader->Program, "cameraPosition"), cameraPos.x, cameraPos.y, cameraPos.z);


	glBindVertexArray(pointsVAO);
	glEnable(GL_VERTEX_PROGRAM_POINT_SIZE);
	glPointParameteri(GL_POINT_SPRITE_COORD_ORIGIN, GL_LOWER_LEFT);
	//glPointSize(50);
	glDrawArrays(GL_POINTS, 0, count);
	glEnable(GL_DEPTH_TEST);


}

