#include <iostream>
#include <thrust/device_vector.h>
#include <chrono>
#include <thread>
#include <math.h>



#include "Common/GpuCommons.h"


#include "Fluid/Fluid_3D_FLIP.cuh"
#include "Fluid/Fluid_3D_PCISPH.cuh"
#include "Fluid/Fluid_3D_PBF.cuh"
#include "Fluid/Fluid_3D.cuh"




#include "Rendering/Renderer3D/camera.h"
#include "Common/InputHandler.h"
#include "Rendering/Renderer3D/PointSprites.h"

#include "Rendering/DrawCommand.h"


#include "Rendering/WindowInfo.h"
#include "Fluid/FluidConfig.cuh"

#include "UI/ui.h"


int main( void ) {

	initOpenGL();

    int screenWidth;
    int screenHeight;

	getScreenDimensions(screenWidth, screenHeight);

	WindowInfo& windowInfo = WindowInfo::instance();
	InputHandler::Handler& inputHandler = InputHandler::Handler::instance();

	windowInfo.windowWidth = screenWidth * 0.9;
	windowInfo.windowHeight = windowInfo.windowWidth / 2;

    GLFWwindow* window = createWindowOpenGL(windowInfo.windowWidth, windowInfo.windowHeight);

	nk_context* uiContext = createUI(window);

    glfwSetKeyCallback(window, InputHandler::key_callback);
    glfwSetCursorPosCallback(window, InputHandler::mouse_callback);

	std::shared_ptr<Camera> camera;
	
	double framesSinceLast = 0;
	double lastSecond = glfwGetTime();
	double lastFrameTime = glfwGetTime();

	glEnable(GL_BLEND);


	FluidConfig config;
	std::shared_ptr<Fluid_3D> fluid;

	

	RenderMode renderMode = RenderMode::Mesh;

	bool paused = true;

	bool hasCreatedFluid = false;

	

    while(!glfwWindowShouldClose(window)){


        glEnable(GL_DEPTH_TEST);
        glDepthFunc(GL_LESS);
        glClear(GL_COLOR_BUFFER_BIT);
        glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
        glClearColor(0,0,0,1);
        glfwPollEvents();

        InputHandler::doMovement();

		if (InputHandler::Handler::instance().keys[GLFW_KEY_SPACE]) {
			InputHandler::Handler::instance().keys[GLFW_KEY_SPACE] = false;
			paused = !paused;
		}
		if (InputHandler::Handler::instance().keys[GLFW_KEY_RIGHT_SHIFT]) {
			InputHandler::Handler::instance().keys[GLFW_KEY_RIGHT_SHIFT] = false;
			renderMode = (RenderMode)(((int)renderMode + 1) % (int)RenderMode::MAX);
		}


        double currentTime = glfwGetTime();

		if (hasCreatedFluid) {

			float near = 0.1;
			float far = 1000;

			glm::mat4 view = camera->getViewMatrix();
			float widthHeightRatio = (float)windowInfo.windowWidth / (float)windowInfo.windowHeight;
			glm::mat4 projection = glm::perspective(camera->FOV, widthHeightRatio, near, far);

			glm::vec3 fluidCenter = fluid->getCenter();

			glm::vec3 lightPos(fluidCenter.x, 30, fluidCenter.y);

			DrawCommand drawCommand = {
			view,projection,camera->position,windowInfo.windowWidth,windowInfo.windowHeight,camera->FOV,near,far,
			renderMode,paused,lightPos
			};

			if (!paused) {
				fluid->simulationStep();
			}

			fluid->draw(drawCommand);
		}

		

		drawUI(uiContext,config, [&]() 
			{
				if (config.initialVolumes.size() == 0) {
					std::cout << "ERROR: No Initial Volumes" << std::endl;
					return;
				}

				if (hasCreatedFluid) {
					fluid.reset();
				}
				if (config.method == "FLIP") {
					fluid = std::static_pointer_cast<Fluid_3D, Fluid_3D_FLIP::Fluid>(std::make_shared<Fluid_3D_FLIP::Fluid>());
				}
				else if (config.method == "PCISPH") {
					fluid = std::static_pointer_cast<Fluid_3D, Fluid_3D_PCISPH::Fluid>(std::make_shared<Fluid_3D_PCISPH::Fluid>());
				}
				else if (config.method == "PBF") {
					fluid = std::static_pointer_cast<Fluid_3D, Fluid_3D_PBF::Fluid>(std::make_shared<Fluid_3D_PBF::Fluid>());
				}
				
				

				fluid->init(config);
				hasCreatedFluid = true;

				camera = std::make_shared<Camera>(fluid->getCenter());
				inputHandler.camera = camera;
			}
		);

        ++framesSinceLast;

        if(currentTime-lastSecond>=1){
            double FPS = (double)framesSinceLast/(currentTime-lastSecond);
            std::cout<<"FPS: "<<FPS<<std::endl;
			std::string fpsText = "Aquarius  " + std::to_string(FPS) + " FPS";
			glfwSetWindowTitle(window, fpsText.c_str());
            lastSecond = currentTime;
            framesSinceLast = 0;
        }

        lastFrameTime = currentTime;

        printGLError();
        glfwSwapBuffers(window);

        //break;
    }

	fluid.reset();
	printGLError();

    std::cout<<"finished everything"<<std::endl;
	hipDeviceReset();

    return 0;
}
